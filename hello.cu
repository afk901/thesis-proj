
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void sayHello() {
  printf("Hello from GPU!\n");
}

int main() {
  printf("Hello from CPU!\n");

  sayHello<<<1,1>>>();
  hipDeviceSynchronize();
}